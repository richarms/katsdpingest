#include "hip/hip_runtime.h"


#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>
#include <sstream>
#include <stdexcept>
#include <vector>
using std::cout;
using std::cerr;
using std::endl;
#include <cstdlib>

#include "gmrt_corr_lib.h"
#include "gmrt_corr_cuda.h"

#include "kernel_defs.hpp"
#include "maths_functions.hpp"
#include "device_interface.hpp"
#include "fill_device.hpp"
#include "unpack_device.hpp"
#include "phase_shift_device.hpp"
#include "fft_device.hpp"
#include "mac_device.hpp"
#include "stopwatch.hpp"


inline void start_profiling(const hipEvent_t& start_event,
                            const hipStream_t& stream=0)
{
#ifdef _PROFILE
  hipEventRecord(start_event, stream);
#endif

}
inline float stop_profiling(const hipEvent_t& start_event,
                            const hipEvent_t& stop_event,
                            const hipStream_t& stream=0)
{
  float elapsed = 0.f;
#ifdef _PROFILE
  hipEventRecord(stop_event, stream);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&elapsed, start_event, stop_event);
#endif
  return elapsed;
}


int gmrt_corr_cuda_create_events(gmrt_corr_profile_t * p)
{
	setDevice(0);
  hipEventCreate(&(p->start_event));
  hipEventCreate(&(p->stop_event));
  p->h2d_time = 0;
  p->d2h_time = 0;
  p->unpack_time = 0;
  p->mac_time = 0;
  p->fft_time = 0;
  p->phase_time = 0;
  p->run_count = 0;
  p->io_time = 0;
	return 0;
}

int gmrt_corr_cuda_alloc(gmrt_corr_data_t * d, gmrt_corr_params_t p)
{

	size_t q;
	for( q=0; q<p.max_queue_count; ++q ) {
    createQueue(d->queues[q]);
    mallocHost(  d->h_raw[q], p.max_raw_word_count * p.ant_count * sizeof(RawWord));
    mallocDevice(d->d_raw[q], p.max_raw_word_count * p.ant_count * sizeof(RawWord));
    mallocDevice(d->d_unpacked[q], p.max_unpacked_count*p.ant_count*sizeof(Real));
  }
  mallocDevice(d->d_output, p.output_count*sizeof(Complex));
  mallocHost(  d->h_output, p.output_count*sizeof(Complex));

	return 0;
}

int gmrt_corr_cuda_dealloc(gmrt_corr_data_t * d, gmrt_corr_params_t p)
{
	freeDevice(d->d_output);
  freeHost(d->h_output);
  for( size_t q=0; q<p.max_queue_count; ++q ) {
    freeDevice(d->d_unpacked[q]); 
    freeDevice(d->d_raw[q]);
    freeHost(d->h_raw[q]);
    destroyQueue(d->queues[q]);
  }
	return 0;
}

float gmrt_corr_cuda_run(gmrt_corr_data_t * d, gmrt_corr_params_t * p, gmrt_corr_profile_t * prof,
                         SourceParType *source, CorrType * corr)
{
  Complex zero;
  zero.x = 0.f;
  zero.y = 0.f;
  
  Stopwatch timer;
  Stopwatch iotimer;
  timer.start();

  // Delay and fringe structs [for delay library]
  //delay_vals_t    delays[NCHAN*NUM_ANT];
  //fringe_vals_t   fringes[NCHAN*NUM_ANT];

  size_t bytes_to_read;
  size_t bytes_read;
  
  bool finished = false;
  // Begin run loop     
  while( !finished ) {
    prof->run_count++;
    if (p->verbose)
    {
      cout << "Beginning new accumulation..." << endl;
      cout << "=============================" << endl;
    }
    // Zero the accumulation result
    fill_device(d->d_output, d->d_output+p->output_count, zero);
    // Iterate over accumulation gulps
    for( size_t accum=0; accum<p->science_accum_size; accum+=p->gulp_accum_size ) {
      // Store gulp size, dealing with incomplete final gulp
      p->gulp_accum_size = math::min(p->gulp_accum_size, p->science_accum_size-accum);
      if (p->verbose)
      {
        cout << "Beginning new gulp accumulating " << p->gulp_accum_size << " cross correlations" << endl;
        cout << "---------------------------------------------------" << endl;
      }
        
      size_t queue_count = 0;
      // Iterate over queues in the gulp
      for( size_t queue=0; queue<p->gulp_accum_size; queue+=p->max_queue_accum_size ) {
        // Store queue's size, dealing with incomplete final queue
        d->queue_accum_size[queue_count] = math::min(p->max_queue_accum_size, p->gulp_accum_size-queue);
        // Calculate the unpack count and raw word count for this queue
        d->unpacked_count[queue_count] = d->queue_accum_size[queue_count] * 2*p->freq_count;
        d->raw_word_count[queue_count] = d->unpacked_count[queue_count] / SAMPLES_PER_WORD;
        if (p->verbose)
        {
          cout << "Using new queue accumulating " << d->queue_accum_size[queue_count] << " cross correlations" << endl;
          cout << "  => " << d->unpacked_count[queue_count] << " samples" << endl;
          cout << "  => " << d->raw_word_count[queue_count] << " raw words" << endl;
        }
        // Track the number of queues used for this gulp
        ++queue_count;       
      }
      if (p->verbose)
      {
        cout << "Total queues used in this gulp: " << queue_count << endl;
        cout << "Beginning data load..." << endl;
      }
      // Load data
      for( size_t q=0; q<queue_count; ++q ) {
        if (p->verbose)
          cout << "Queue " << q << " reading data from source..." << endl;
        iotimer.start();
        for( size_t ant=0; ant<p->ant_count; ++ant ) {
          // Read data from source
          bytes_to_read = d->raw_word_count[q];
          //bytes_to_read = d->raw_word_count[q]*sizeof(RawWord);
          bytes_read = d->io_function(d, ant, (char *) (d->h_raw[q] + ant*d->raw_word_count[q]), bytes_to_read);
          //fprintf(stderr, "ant=%d, bytes_to_read=%d\n", ant, bytes_to_read);
          //bytes_read = d->io_function(d, ant, (char *) (d->h_raw[q]+ant*d->raw_word_count[q]), bytes_to_read);
          if (bytes_to_read != bytes_read) 
          {
            if (p->verbose)
              cerr << "EOD detected maybe??" << endl;
            finished = true;
          }
        }
        iotimer.stop();
        if (p->verbose)
          cout << "Queue " << q << " copying data to device..." << endl;
        // Copy to device
        start_profiling(prof->start_event, d->queues[q]);
        copyHostToDevice(d->d_raw[q], d->h_raw[q],
                         d->raw_word_count[q]*p->ant_count*sizeof(RawWord),
                         d->queues[q]);
        prof->h2d_time += stop_profiling(prof->start_event, prof->stop_event, d->queues[q]);
      }
      if (p->verbose)
        cout << "Beginning compute..." << endl;
      // Compute
      for( size_t q=0; q<queue_count; ++q ) {
        if (p->verbose)
          cout << "Queue " << q << " unpacking data..." << endl;
        // Unpack
        start_profiling(prof->start_event, d->queues[q]);
        // TODO: Check this!
        /* // This unpacks antenna-interlaced data from the IBOB
        unpack_to_real_ibob_device((unsigned int*)d->d_raw[q],
                              (float4*)d->d_unpacked[q],
                              d->raw_word_count[q],
                              p->ant_count,
                              math::log2(p->ant_count),
                              d->unpacked_count[q]);
        */
        // This unpacks separate time series from each antenna
        //unpack_to_complex_device(d->d_raw[q], d->d_unpacked[q],
        //                         d->raw_word_count[q]*p->ant_count, d->queues[q]);
        unpack_to_real_device(d->d_raw[q], d->d_unpacked[q], 
                              d->raw_word_count[q]*p->ant_count,
                              sizeof(RawWord)*8/SAMPLES_PER_WORD,
                              d->queues[q]);
        
        /*
        // HACK to write unpacked first antenna to file
        copyDeviceToHost(h_unpacked[q], d->d_unpacked[q],
                         d->unpacked_count[q]*sizeof(Real),
                         d->queues[q]);
        synchronize(d->queues[q]);
        std::ofstream upk_file("unpacked_0.dat");
        for( size_t u=0; u<d->unpacked_count[q]; ++u ) {
          upk_file << h_unpacked[q][u] << "\n";
        }
        upk_file.close();
        */
        prof->unpack_time += stop_profiling(prof->start_event, prof->stop_event, d->queues[q]);
       
        if (p->verbose)
        {  
          cout << "Queue " << q << " FFTing data" << endl;
          cout << "  batch: " << d->queue_accum_size[q] * p->ant_count << endl;        
        }
        // FFT
        //if( d->queue_accum_size[q] != p->max_queue_accum_size ) {
        //  cout << "**** ERROR: Need to implement on-the-fly fft plan creation!" << endl;  
        //}
        start_profiling(prof->start_event, d->queues[q]);
        fft_r2c_short_device(d->d_unpacked[q], (Complex*)d->d_unpacked[q],
                             2*p->freq_count,
                             d->queue_accum_size[q] * p->ant_count,
                             d->queues[q]);
        prof->fft_time += stop_profiling(prof->start_event, prof->stop_event, d->queues[q]);
        
        // Phase shift
        /*
        start_profiling(prof->start_event, d->queues[q]);
       
        // need to calculate the fractional delay and apply  
        phase_shift_device((Complex*)d->d_unpacked[q],
                           (Complex*)d->d_unpacked[q],
                           d->unpacked_count[q]/2 * p->ant_count,
                           0.1f, // TODO: Insert delay here
                           d->queues[q]);
        prof->phase_time += stop_profiling(prof->start_event, prof->stop_event, d->queues[q]);
        */
        if (p->verbose)
          cout << "Queue " << q << " MACing data..." << endl;
        // MAC
        start_profiling(prof->start_event, d->queues[q]);
        size_t in_stride1  = p->freq_count;
        size_t in_stride2  = d->unpacked_count[q] / 2; // Complex samples
        size_t out_stride1 = p->freq_count;
        size_t out_stride2 = p->ant_count * out_stride1;
        mac_device((Complex*)d->d_unpacked[q], in_stride1, in_stride2,
                   p->ant_count, p->freq_count, d->queue_accum_size[q],
                   d->d_output, out_stride1, out_stride2, p->do_full_polar,
                   d->queues[q]);
        
        // HACK for testing half_floats
        /*mac_device((ushort2*)d->d_unpacked[q], in_stride1, in_stride2,
                   p->ant_count, p->freq_count, d->queue_accum_size[q],
                   (ushort2*)d_output, out_stride1, out_stride2, p->do_full_polar,
                   d->queues[q]);
        */
        prof->mac_time += stop_profiling(prof->start_event, prof->stop_event, d->queues[q]);
      }
    } // End of iteration over accumulation gulps
    
    if (p->verbose)
    {
      cout << "===========================" << endl;
      cout << "End of accumulation reached" << endl;
    }
      
    // TODO: Could compute second half of spectrum here before
    //       copying back to host.
   
    if (p->verbose) 
      cout << "Retrieving results from device..." << endl;
    // Copy back to host
    start_profiling(prof->start_event);
    copyDeviceToHost(d->h_output, d->d_output,
                     p->output_count*sizeof(Complex));
    prof->d2h_time += stop_profiling(prof->start_event, prof->stop_event);
      
    if (p->verbose)
      cout << "Writing output to destination..." << endl;
    // Output
    synchronize(); // Wait for device->host copy to finish

    if (p->write_output && !finished) {
      // HACK to avoid waiting ages when benchmarking
      if( p->ant_count <= 16 ) {
        size_t ij = 0;
        for( size_t i=0; i<p->ant_count; ++i ) {
          // Note: This is a little trick to divide by either 1 (single-pol)
          //       or 2 (full-polar) when skipping terms near the diagonal.
          size_t b = p->do_full_polar+1;
          for( size_t j=i/b*b; j<p->ant_count; ++j ) {
            ++ij;
            // Open the output destinations
            std::stringstream ss1, ss2;
            //ss1 << "spec_" << ij;
            ss1 << "spec_" << i << "_" << j;
            std::ofstream spectrum_out(ss1.str().c_str());
            spectrum_out.precision(5); // sig figs
            //ss2 << "phase_" << ij;
            ss2 << "phase_" << i << "_" << j;
            std::ofstream phase_out(ss2.str().c_str());
            phase_out.precision(5); // sig figs
            // Iterate over each frequency channel
            for( size_t f=0; f<p->freq_count; ++f ) {
              Complex cij = d->h_output[f+p->freq_count*(i+j*p->ant_count)];
              // Normalise the accumulations
              cij.x /= p->science_accum_size;
              cij.y /= p->science_accum_size;
              // Write the amplitude and phase
              spectrum_out << sqrt(cij.x*cij.x + cij.y*cij.y) << endl;
              // HACK TODO: Work out why this is needed to match old results
              cij.y *= -1;
              phase_out    << atan2(cij.y, cij.x) << endl;
            }
          }
        }
      }
    }
    // TODO: Remove this to run continuously
    //finished = true;

  } // End of run loop
  prof->io_time = iotimer.getTime() * 1000;
  timer.stop();

  return timer.getTime();
}
